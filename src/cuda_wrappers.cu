 // wrappers file
// compile with:
// rm cuda_wrappers.o; nvcc -lcublas -lcurand -arch=sm_70 src/cuda_wrappers.cu -c;
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <assert.h>
#include <hip/hip_complex.h>

#define tpb 128

/*__device__ double atomicDoubleAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}*/


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void vect_dble(double *a, int N)
{
   int idx = threadIdx.x+blockIdx.x*gridDim.x;
   if (idx<N)printf(" %lf \n", a[idx]);
}

extern "C" void cuda_malloc_double(void **a_d, int Np)
{
   // Allocate memory on GPU
   //double **loc_a_d=(double **) a_d;
   //printf(" malloc double \n" );
   //if ( cudaSuccess != cudaMalloc( loc_a_d, sizeof(double) * Np )); exit(0);
   //gpuErrchk(cudaMalloc( a_d, sizeof(double) * Np ));
   //printf("Error in malloc double \n" );
   /*if ( cudaSuccess != cudaMalloc( a_d, sizeof(double) * Np ));
   printf("Error in malloc double \n" );*/
   hipMallocAsync( a_d, sizeof(double) * Np,0);
   return;
}

extern "C" void cuda_malloc_double_complex(void **a_d, int Np)
{
   // Allocate memory on GPU
   //double **loc_a_d=(double **) a_d;
   //printf(" malloc double \n" );
   //if ( cudaSuccess != cudaMalloc( loc_a_d, sizeof(double) * Np )); exit(0);
   //gpuErrchk(cudaMalloc( a_d, sizeof(double) * Np ));
   //printf("Error in malloc double \n" );
   /*if ( cudaSuccess != cudaMalloc( a_d, sizeof(double) * Np ));
   printf("Error in malloc double \n" );*/
   hipMallocAsync( a_d, sizeof(hipDoubleComplex) * Np,0);
   return;
}

extern "C" void cuda_malloc_int(int **a_d, int Np)
{
   // Allocate memory on GPU
   //gpuErrchk(cudaMalloc( a_d, sizeof(int) * Np ));
   hipMallocAsync( a_d, sizeof(int) * Np, 0 );
   return;
}


extern "C" void cuda_free(double **a_d)
{
   hipFree(*a_d);
   //printf("GPU memory freed \n");
   return;
}

extern "C" void GPU_fill_rand(double *A, int N, int ccc) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock() + (unsigned long long)  ccc * N);

	// Fill the array with random numbers on the device
	hiprandGenerateUniformDouble(prng, A,N);
  //vect_dble<<<(N+128-1)/128,128>>>(A,N);
  //cudaDeviceSynchronize();
  printf("\n Filled \n");
}

extern "C" void cuda_cpy_double_htod(double *a, double *a_d, int N)
{
   //cudaMemcpy(a_d, a, sizeof(double) * N, cudaMemcpyHostToDevice);
   hipMemcpyAsync(a_d, a, sizeof(double) * N, hipMemcpyHostToDevice);


   return;
}

extern "C" void cuda_cpy_double_complex_htod(double *a, double *a_d, int N)
{
   //cudaMemcpy(a_d, a, sizeof(double) * N, cudaMemcpyHostToDevice);
   hipMemcpyAsync(a_d, a, sizeof(hipDoubleComplex) * N, hipMemcpyHostToDevice);


   return;
}



extern "C" void cuda_cpy_int_htod(int *a, int *a_d, int N)
{

   //cudaMemcpy(a_d, a, sizeof(int) * N, cudaMemcpyHostToDevice );
   hipMemcpyAsync(a_d, a, sizeof(int) * N, hipMemcpyHostToDevice );
   return;
}


extern "C" void cuda_cpy_double_dtoh(double *a_d, double *a ,int N)
{
  //cudaMemcpyAsync( a, a_d, sizeof(double) * N, cudaMemcpyDeviceToHost );
  hipMemcpy( a, a_d, sizeof(double) * N, hipMemcpyDeviceToHost );
  //gpuErrchk(cudaMemcpy( a, a_d, sizeof(double) * N, cudaMemcpyDeviceToHost ));
   //printf("\nTest cpy D to H \n");
   
   return;
}
extern "C" void cuda_cpy_double_dtod(double *b_d, double *a_d,int N)
{
   hipMemcpyAsync( a_d, b_d, sizeof(double) * N, hipMemcpyDeviceToDevice );
   //cudaMemcpy( a_d, b_d, sizeof(double) * N, cudaMemcpyDeviceToDevice );

   return;
}


extern "C" void create_cublas_handle(hipblasHandle_t *handle)
{
 	 hipblasCreate(handle);
   //printf("\n cublas handle created \n");

   return;
}

extern "C" void destroy_cublas_handle(hipblasHandle_t *handle)
{
 	 // Destroy the handle
   hipblasDestroy(*handle);
   //printf("\n cublas handle destroyed \n");

   return;
}

__global__ void gpu_pow(double *a,double *b, double zeta, int N)
{
   int idx = threadIdx.x+blockIdx.x*blockDim.x;
   if (idx<N){
   double loca=a[idx];
   b[idx]=pow(loca,zeta);

 }
}

extern "C" void gpu_kernels_pow(double *a,double *b, double zeta, int size)
{
  int ntpb=256;
  int nblocks=(size+ntpb-1)/ntpb;
  gpu_pow<<<nblocks,ntpb>>>(a,b,zeta, size);
  //gpuErrchk( cudaPeekAtLastError() );
  //gpuErrchk( cudaDeviceSynchronize() );
  return;
}

extern "C" void gpu_blas_mmul_t_n(hipblasHandle_t handle, const double *Qs_d, const double *soap_d, double *kernels_d, const int n_sparse, const int n_soap, const int n_sites)
//                                                           const double *A,     const double *B,         double *C,       const int nAx,
// const int nAy,      const int nBy,double *b, double zeta, int N)
{
// (cublasHandle_t handle, const double *Qs_d, const double *soap_d, double *kernels_d, const int n_sparse, const int n_soap, const int n_sites,double *b, double zeta, int N)
	const double alf = 1;
	const double bet = 0;

// soap(n_soap,n_sites)
// Qs(1:n_soap, 1:n_sparse)
// kernels(1:n_sites, 1:n_sparse)
// call dgemm( "t", "n", n_sites, n_sparse, n_soap, 1.d0, soap, n_soap, Qs, n_soap, 0.d0, kernels, n_sites)

	// Do the actual multiplication
  //printf("\n Pongo Longo \n");
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n_sites, n_sparse, n_soap, &alf, soap_d, n_soap, Qs_d, n_soap, &bet, kernels_d, n_sites);

  return;
}

extern "C" void gpu_blas_mvmul_n(hipblasHandle_t handle,  double *kernels_copy_d, const double *alphas_d, double *energies_d, const int n_sites, const int n_sparse)
{

	const double alf = 1;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;

	// Do the actual multiplication
  hipblasDgemv(handle, HIPBLAS_OP_N, n_sites,n_sparse, alpha, kernels_copy_d, n_sites, alphas_d, 1, beta, energies_d, 1);
 return;
}



__global__ void gpu_simpleaxpc(double *a, double dccc, double e0, int N)
{
   int idx = threadIdx.x+blockIdx.x*blockDim.x;
   if (idx<N){
   double loca=a[idx];
   a[idx]=dccc*loca+e0;
 }
}

extern "C" void gpu_axpc(double *a, double dccc, double e0, int size)
{
  int ntpb=256;
  int nblocks=(size+ntpb-1)/ntpb;
  gpu_simpleaxpc<<<nblocks,ntpb>>>(a,dccc,e0, size);
  /*gpuErrchk( cudaPeekAtLastError() );
  gpuErrchk( cudaDeviceSynchronize() );*/
  return;

}

extern "C" void wrappers_all(double *soap, double *kernels, double *kernels_copy, double *Qs, double *energies, double delta, double zeta, double e0, int n_sites, int n_soap, int n_sparse, int size_kernels, int size_soap, int size_Qs, int size_alphas, int  size_energies)
{
  int ntpb=256;
  int nblocks=(size_kernels+ntpb-1)/ntpb;
  // Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);
  double *kernels_d, *kernels_copy_d, *soap_d, *Qs_d, *energies_d;
  hipMalloc( &kernels_d, sizeof(double) * size_kernels );
  hipMalloc( &kernels_copy_d, sizeof(double) * size_kernels );
  hipMalloc( &soap_d, sizeof(double) * size_soap );
  hipMalloc( &Qs_d, sizeof(double) * size_Qs );
  hipMalloc( &energies_d, sizeof(double)*size_energies);


  const double alf = 1;
  const double bet = 0;

  hipMemcpy(kernels_d, kernels, sizeof(double) * size_kernels, hipMemcpyHostToDevice );
  hipMemcpy(soap_d, soap, sizeof(double) * size_soap, hipMemcpyHostToDevice );
  hipMemcpy(Qs_d, Qs, sizeof(double) * size_Qs, hipMemcpyHostToDevice );
  // Do the actual multiplication

  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n_sites, n_sparse, n_soap, &alf, soap_d, n_soap, Qs_d, n_soap, &bet, kernels_d, n_sites);
//cublasDgemm(handle, CUBLAS_OP_T, CUBLAS_OP_N,  nBy, nAx, nAy, alpha, B, nAy, A, nAy, beta, C, nBy);
    //printf("\n cublasDgemm \n");
  // gpu_blas_mmul_t_n(cubhandle,     A,     B,      C,         nAx,      nAy,       nBy,             bb, zeta, N)
  // gpu_blas_mmul_t_n(cublas_handle, Qs_d, soap_d, kernels_d, n_sparse, n_soap, n_sites, kernels_copy_d, zeta, size_kernels)

  hipMemcpy( kernels, kernels_d, sizeof(double) * size_kernels, hipMemcpyDeviceToHost );
  gpu_pow<<<dim3(nblocks,1,1),dim3(ntpb,1,1)>>>(kernels_d,kernels_copy_d, zeta, size_kernels);
  hipMemcpy( kernels_copy, kernels_copy_d, sizeof(double) * size_kernels, hipMemcpyDeviceToHost );
	// Destroy the handle
	hipblasDestroy(handle);
  hipFree(kernels_d);
  hipFree(kernels_copy_d);
  hipFree(soap_d);
  hipFree(Qs_d);
  hipFree(energies_d);
  //printf("\n %d %d %d %d %d %d %d %d  \n", n_sites, n_soap, n_sparse, size_kernels,  size_soap,  size_Qs,  size_alphas,  size_energies);
  //printf("\n %d %d %d\n", nblocks,ntpb, size_kernels);
  //exit(0);
 return;
}

extern "C" void cuda_set_device( int my_rank)
{

  int  num_gpus=0;
  gpuErrchk(hipGetDeviceCount(&num_gpus));
  gpuErrchk(hipSetDevice(my_rank%num_gpus));
  return;
}


__global__ void matvect_kernels(double *kernels_d, double *alphas_d,int  n_sites, int n_sparse)
{
   int idx = threadIdx.x+blockIdx.x*blockDim.x;
   int ispa=idx/n_sites;
   int isite=idx%n_sites;
   if (ispa<n_sparse && isite<n_sites){
     double lock=kernels_d[idx]*alphas_d[ispa];
     kernels_d[idx]=lock;
 }
}

extern "C" void cuda_matvect_kernels(double *kernels_d, double *alphas_d,int  n_sites, int n_sparse)
{
  int  ntpb=256;
  int nblocks=(n_sites*n_sparse+ntpb-1)/ntpb;
  matvect_kernels<<<nblocks,ntpb>>>(kernels_d,alphas_d,n_sites,n_sparse);
  /*gpuErrchk( cudaPeekAtLastError() );
  gpuErrchk( cudaDeviceSynchronize() );*/
  return;
}



__global__ void matvect_qs(double *qs_d,double *qs_copy_d, double *alphas_d,int  n_soap, int n_sparse)
{
   int idx = threadIdx.x+blockIdx.x*blockDim.x;
   int ispa=idx/n_soap;
   int isoap=idx%n_soap;
   if (ispa<n_sparse && isoap<n_soap){
     double lock=qs_d[idx]*alphas_d[ispa];
     qs_copy_d[idx]=lock;
 }
}

extern "C" void cuda_matvect_qs(double *qs_d,double *qs_copy_d, double *alphas_d,int  n_soap, int n_sparse)
{
  /*
  alphas(n_sparse)
  allocate( Qs_copy(1:n_soap, 1:n_sparse) )
  do i = 1, n_soap
    Qs_copy(i,:) = Qs(i,:)*alphas(:)
  end do
  */
  int  ntpb=256;
  int nblocks=(n_soap*n_sparse+ntpb-1)/ntpb;
  matvect_qs<<<nblocks,ntpb>>>(qs_d,qs_copy_d,alphas_d,n_soap,n_sparse);
  /*gpuErrchk( cudaPeekAtLastError() );
  gpuErrchk( cudaDeviceSynchronize() );*/
  return;
}


// gpu_blas_mmul_n_t(cubhandle, kernels_der_d, Qs_copy_d, Qss_d, n_sparse, n_soap, n_sites, cdelta)
extern "C" void gpu_blas_mmul_n_t(hipblasHandle_t handle, const double *kernels_der_d, const double *Qs_copy_d, double *Qss_d, const int n_sparse, const int n_soap, const int n_sites, double cdelta)
{

	const double alf = cdelta;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;
// soap(n_soap,n_sites)
// Qs(1:n_soap, 1:n_sparse)
// kernels(1:n_sites, 1:n_sparse)
// call dgemm( "t", "n", n_sites, n_sparse, n_soap, 1.d0, soap, n_soap, Qs, n_soap, 0.d0, kernels, n_sites)
// cublasDgemm(handle, CUBLAS_OP_T, CUBLAS_OP_N, n_sites, n_sparse, n_soap, alpha, soap_d, n_soap, Qs_d, n_soap, beta, kernels_d, n_sites);

// allocate( kernels_der(1:n_sites, 1:n_sparse)
// allocate( Qs_copy(1:n_soap, 1:n_sparse) ))
// allocate( Qss(1:n_sites, 1:n_soap) )
// call dgemm("n", "t", n_sites, n_soap, n_sparse, cdelta, kernels_der, n_sites, Qs_copy, n_soap, 0.d0, Qss, n_sites)
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n_sites,  n_soap, n_sparse, alpha,  kernels_der_d, n_sites,Qs_copy_d, n_soap, beta, Qss_d, n_sites);
}

/*
!!$OMP parallel do private(i,j,l,j2,this_Qss)
!      l = 0
      do i = 1, n_sites
        this_Qss = Qss(i,1:n_soap)
!        do j = 1, n_neigh(i)
!          l = l + 1
         do l = neighbors_beg(i), neighbors_end(i)
          j2 = mod(neighbors_list(l)-1, n_sites0) + 1
          do k = 1, 3
            this_force(k) = dot_product(this_Qss, soap_der(k,:,l))
            forces(k, j2) = forces(k, j2) + this_force(k)
          end do
!         This is a many body potential, so there's no factor of 1/2 here
!          virial = virial + dot_product( this_force(1:3), xyz(1:3,l) )
          do k1 = 1, 3
            do k2 =1, 3
              virial(k1, k2) = virial(k1, k2) + 0.5d0 * (this_force(k1)*xyz(k2,l) + this_force(k2)*xyz(k1,l))
            end do
          end do
        end do
      end do
!!$OMP end parallel do
*/
__global__ void cuda_soap_forces_virial(int *n_neigh_d,int n_sites,
                                        double *Qss_d,int n_soap,
                                        int *neighbors_beg_d,
                                        double3 *soap_der_d,
                                        double3 *xyz_d, double *virial_d,
                                        int *neighbors_list_d,int n_sites0, double *forces_d)
{
  int i_site=blockIdx.x;
  int i_nn=blockIdx.y;
  int n_nn_i_site=n_neigh_d[i_site];
  int tid=threadIdx.x;

  if(i_nn<n_nn_i_site)  // only the blocks with index smaller than the number of neighbors are participating in the calcultions
  {
    int l_nn=neighbors_beg_d[i_site]-1+i_nn;


    __shared__ double shxthis_block_force[tpb];
    __shared__ double shythis_block_force[tpb];
    __shared__ double shzthis_block_force[tpb];

    shxthis_block_force[tid]=0;
    shythis_block_force[tid]=0;
    shzthis_block_force[tid]=0;
    double locx_this_force=0;
    double locy_this_force=0;
    double locz_this_force=0;


    for(int ii=tid; ii< n_soap;ii=ii+tpb)
    {
        int i_Qss=i_site+ii*n_sites;
        double loc_this_Qss=Qss_d[i_Qss];// this rea<d  seems OK
        int in_soap_der=(l_nn*n_soap+ii);
        double3 loc_soap_der=soap_der_d[in_soap_der];
        locx_this_force+=loc_this_Qss*loc_soap_der.x;
        locy_this_force+=loc_this_Qss*loc_soap_der.y;
        locz_this_force+=loc_this_Qss*loc_soap_der.z;
      }

    shxthis_block_force[tid]=locx_this_force;
    shythis_block_force[tid]=locy_this_force;
    shzthis_block_force[tid]=locz_this_force;

    __syncthreads();

    //reduction
    for (int s=tpb/2; s>0; s>>=1) // s=s/2
    {
      if (tid < s)
      {
        shxthis_block_force[tid] +=shxthis_block_force[tid + s];
        shythis_block_force[tid] +=shythis_block_force[tid + s];
        shzthis_block_force[tid] +=shzthis_block_force[tid + s];
      }
      __syncthreads();

    }
    //  at this point this_force is computed
    if(tid==0)
    {
      int j2=(neighbors_list_d[l_nn]-1) % (n_sites0);
      if(j2>= n_sites0)
      {printf("j2 the error! \n");}

      atomicAdd(&forces_d[j2*3]  , shxthis_block_force[0]);
      atomicAdd(&forces_d[j2*3+1], shythis_block_force[0]);
      atomicAdd(&forces_d[j2*3+2], shzthis_block_force[0]);

      // now the virial
      double this_force[3];
      this_force[0]=shxthis_block_force[0];
      this_force[1]=shythis_block_force[0];
      this_force[2]=shzthis_block_force[0];

      double3 tmp_xyz;
      tmp_xyz=xyz_d[l_nn];
      double this_xyz[3];
      this_xyz[0]=tmp_xyz.x;
      this_xyz[1]=tmp_xyz.y;
      this_xyz[2]=tmp_xyz.z;

      for(int k1=0;k1<3;k1++)
      {
        for(int k2=0;k2<3;k2++)
        {
          double loc_viri=0.5*(this_force[k1]*this_xyz[k2]+this_force[k2]*this_xyz[k1]);
          atomicAdd(&virial_d[k2+3*k1], loc_viri);
        }
      }
    }
  }
}

extern "C" void gpu_final_soap_forces_virial(int *n_neigh_d, int n_sites, int maxnn,
                                             double *Qss_d,int n_soap, int *neighbors_beg_d, double3 *soap_der_d,
                                             double3 *xyz_d, double *virial_d,
                                             int *neighbors_list_d,int n_sites0, double *forces_d)
{

     dim3 nblocks(n_sites,maxnn,1);
     hipMemsetAsync(forces_d,0, 3*n_sites0*sizeof(double));
     hipMemsetAsync(virial_d,0, 9*sizeof(double));
     cuda_soap_forces_virial<<<nblocks,tpb>>>(n_neigh_d,n_sites,
                                              Qss_d,n_soap, neighbors_beg_d,
                                              soap_der_d, xyz_d, virial_d,
                                              neighbors_list_d, n_sites0, forces_d);

     /*gpuErrchk( cudaPeekAtLastError() );
     gpuErrchk( cudaDeviceSynchronize() );*/

     return;
}


extern "C" void gpu_soap_energies_forces_virial(int *n_neigh_d, int n_sites, int maxnn,
                                             double *Qss_d,int n_soap, int *neighbors_beg_d,
                                             double3 *soap_der_d,
                                             double3 *xyz_d, double *virial_d,
                                             int *neighbors_list_d,int n_sites0, double *forces_d,
                                             hipblasHandle_t handle, double *kernels_der_d, double *Qs_copy_d,
                                             const int n_sparse, double cdelta_force,
                                             double *alphas_d,
                                             double *kernels_d, double mzetam, int size_kernels,
                                             int do_forces,
                                             double *energies_d, double cdelta_ene, double e0, int size_energies,
                                             double *Qs_d, int size_Qs,
                                             double  *kernels_copy_d,
                                             double zeta,
                                             double *soap_d )
{
  gpu_blas_mmul_t_n(handle, Qs_d, soap_d, kernels_d, n_sparse, n_soap, n_sites);
  gpu_kernels_pow( kernels_d, kernels_copy_d,zeta, size_kernels);
  gpu_blas_mvmul_n(handle, kernels_copy_d, alphas_d, energies_d, n_sites, n_sparse);
  gpu_axpc( energies_d,cdelta_ene,e0, size_energies);
  if(do_forces==1)
  {
    cuda_cpy_double_dtod(Qs_d,   Qs_copy_d ,size_Qs);

    gpu_kernels_pow(kernels_d, kernels_der_d,mzetam, size_kernels);
    if(n_sites<n_soap)
    {
      cuda_matvect_kernels(kernels_der_d, alphas_d, n_sites, n_sparse);
    }
    else
    {
     cuda_matvect_kernels(Qs_copy_d, alphas_d, n_soap, n_sparse);
    }
       gpu_blas_mmul_n_t(handle, kernels_der_d, Qs_copy_d, Qss_d, n_sparse,n_soap, n_sites, cdelta_force);

       gpu_final_soap_forces_virial(n_neigh_d, n_sites, maxnn,
                                     Qss_d, n_soap,neighbors_beg_d,
                                     soap_der_d, xyz_d, virial_d,
                                     neighbors_list_d, n_sites0, forces_d);
       /*dim3 nblocks(n_sites,maxnn,1);*/
       /*gpuErrchk(cudaMemset(forces_d,0, 3*n_sites0*sizeof(double)));
       gpuErrchk(cudaMemset(virial_d,0, 9*sizeof(double)));*/

       /*cudaMemsetAsync(forces_d,0, 3*n_sites0*sizeof(double));
       cudaMemsetAsync(virial_d,0, 9*sizeof(double));

       cuda_soap_forces_virial<<<nblocks,tpb>>>(n_neigh_d,n_sites,
                                            Qss_d,n_soap,neighbors_beg_d,
                                            soap_der_d,
                                            xyz_d, virial_d,
                                            neighbors_list_d, n_sites0, forces_d);*/
  }

     /*gpuErrchk( cudaPeekAtLastError() );
     gpuErrchk( cudaDeviceSynchronize() );*/
     return;
}



__global__ void cuda_get_soap_p(double *soap_d, double *multiplicity_array_d, 
                           hipDoubleComplex *cnk_d, int *skip_soap_component_d,
                           int n_sites, int n_soap, int n_max, int l_max)
{
   int i_site = threadIdx.x+blockIdx.x*blockDim.x;
   int k_max=1+l_max*(l_max+1)/2+l_max;
   if (i_site<n_sites){ 
    int counter=0;
    int counter2=0; 
    for(int n=0;n<n_max;n++){
      for(int np=n;np<n_max;np++){
        for(int l=0;l<=l_max;l++){
          if(skip_soap_component_d[l+(l_max+1)*(np+n*n_max)]!=0){
            counter++;
            for(int m=0;m<=l; m++){
              int k=1+l*(l+1)/2+m;
              counter2++;
              hipDoubleComplex tmp_cnk_d=cnk_d[k+k_max*(n+i_site*n_max)];
              soap_d[counter-1+i_site*n_soap]+=multiplicity_array_d[counter2-1]*tmp_cnk_d.x*tmp_cnk_d.y;
            }

          }
        }
      }
    }
 }
}

__global__ void cuda_get_sqrt_dot_p(double *soap_d, double *sqrt_dot_p_d,  
                                int n_sites, int n_soap)
{
  int i_site=blockIdx.x;
  int tid=threadIdx.x;
  __shared__ double sh_sqrt_dot_p_d[tpb];
  double this_dotprod=0.0;
  for(int s=0;s<n_soap;s=s+tpb){
    if(s<n_soap){
      this_dotprod+=soap_d[tid+s*tpb+i_site*n_soap]*soap_d[tid+s*tpb+i_site*n_soap];
    } 
  }
  sh_sqrt_dot_p_d[tid]=this_dotprod;
  __syncthreads();

  //reduction
  for (int s=tpb/2; s>0; s>>=1) // s=s/2
  {
    if (tid < s)
    {
      sh_sqrt_dot_p_d[tid] +=sh_sqrt_dot_p_d[tid + s];
    }
    __syncthreads();

  }
  //  at this point this_force is computed
  if(tid==0){
    double final_dotprod=1.0;
    if(sh_sqrt_dot_p_d[0]>=1.0e-5){
      final_dotprod=sh_sqrt_dot_p_d[0];
    }
    sqrt_dot_p_d[i_site]=final_dotprod;
  }
}

extern "C" void gpu_get_sqrt_dot_p(double *sqrt_dot_d, double *soap_d, double *multiplicity_array_d, 
                                   hipDoubleComplex *cnk_d, int *skip_soap_component_d, 
                                   int n_sites, int n_soap, int n_max, int l_max)
{
  dim3 nblocks=dim3((n_sites+1-tpb)/tpb,1,1);
  dim3 nthreads=dim3(tpb,1,1);
  cuda_get_soap_p<<<nblocks, nthreads>>>(soap_d, multiplicity_array_d, cnk_d, skip_soap_component_d, 
                                         n_sites, n_soap, n_max, l_max);
  cuda_get_sqrt_dot_p<<<n_sites,tpb>>>(soap_d,sqrt_dot_d, n_sites, n_soap);                                      
  return;
}